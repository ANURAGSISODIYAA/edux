
#include <hip/hip_runtime.h>
extern "C"
__global__ void matrixMultiply(double *A, double *B, double *C, int numARows, int numAColumns, int numBColumns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numARows && col < numBColumns) {
        double sum = 0;
        for (int i = 0; i < numAColumns; ++i) {
            sum += A[row * numAColumns + i] * B[i * numBColumns + col];
        }
        C[row * numBColumns + col] = sum;
    }
}
